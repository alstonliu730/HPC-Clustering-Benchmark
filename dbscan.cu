#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <vector>
#include <math.h>

// Structure to store 3D points
struct Point3D {
    float x, y, z;
};

// Constants for DBSCAN
const int NOISE = -1;
const int UNCLASSIFIED = -2;

// Kernel to calculate distance matrix between points
__global__ void calculateDistanceMatrix(Point3D* points, int numPoints, float* distMatrix, float eps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < numPoints && idy < numPoints) {
        float dx = points[idx].x - points[idy].x;
        float dy = points[idx].y - points[idy].y;
        float dz = points[idx].z - points[idy].z;
        
        float dist = sqrt(dx*dx + dy*dy + dz*dz);
        
        // Store 1 if points are within eps distance, 0 otherwise
        distMatrix[idx * numPoints + idy] = (dist <= eps) ? 1.0f : 0.0f;
    }
}

// Kernel to count neighbors for each point
__global__ void countNeighbors(float* distMatrix, int numPoints, int* neighborCounts) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numPoints) {
        int count = 0;
        for (int j = 0; j < numPoints; j++) {
            if (distMatrix[idx * numPoints + j] > 0) {
                count++;
            }
        }
        neighborCounts[idx] = count;
    }
}

// Kernel to expand clusters
__global__ void expandCluster(float* distMatrix, int numPoints, int* labels, 
                            int* neighborCounts, int minPts, int currentCluster, 
                            bool* changed, int* borderPoints, int borderPointsCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < borderPointsCount) {
        int pointIdx = borderPoints[idx];
        
        // Skip already processed points
        if (labels[pointIdx] != currentCluster) {
            return;
        }
        
        for (int j = 0; j < numPoints; j++) {
            if (distMatrix[pointIdx * numPoints + j] > 0 && labels[j] == UNCLASSIFIED) {
                // Mark as part of the current cluster
                atomicExch(&labels[j], currentCluster);
                *changed = true;
                
                // If core point, add to border points for next iteration
                if (neighborCounts[j] >= minPts) {
                    // We'll handle this point in the next iteration
                }
            }
        }
    }
}

// Kernel to find initial core points and mark them with their cluster IDs
__global__ void findCorePoints(int* neighborCounts, int numPoints, int minPts, int* labels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numPoints) {
        if (neighborCounts[idx] >= minPts && labels[idx] == UNCLASSIFIED) {
            // This is a core point that hasn't been assigned to a cluster yet
        }
    }
}

// Host function to run DBSCAN
void dbscan(Point3D* h_points, int numPoints, float eps, int minPts, int* h_labels) {
    // Allocate device memory
    Point3D* d_points;
    float* d_distMatrix;
    int* d_labels;
    int* d_neighborCounts;
    
    hipMalloc((void**)&d_points, numPoints * sizeof(Point3D));
    hipMalloc((void**)&d_distMatrix, numPoints * numPoints * sizeof(float));
    hipMalloc((void**)&d_labels, numPoints * sizeof(int));
    hipMalloc((void**)&d_neighborCounts, numPoints * sizeof(int));
    
    // Initialize all points as UNCLASSIFIED
    for (int i = 0; i < numPoints; i++) {
        h_labels[i] = UNCLASSIFIED;
    }
    
    // Copy data to device
    hipMemcpy(d_points, h_points, numPoints * sizeof(Point3D), hipMemcpyHostToDevice);
    hipMemcpy(d_labels, h_labels, numPoints * sizeof(int), hipMemcpyHostToDevice);
    
    // Calculate distance matrix
    dim3 blockDim(16, 16);
    dim3 gridDim((numPoints + blockDim.x - 1) / blockDim.x, 
                (numPoints + blockDim.y - 1) / blockDim.y);
    
    calculateDistanceMatrix<<<gridDim, blockDim>>>(d_points, numPoints, d_distMatrix, eps);
    hipDeviceSynchronize();
    
    // Count neighbors for each point
    int threadsPerBlock = 256;
    int blocksPerGrid = (numPoints + threadsPerBlock - 1) / threadsPerBlock;
    
    countNeighbors<<<blocksPerGrid, threadsPerBlock>>>(d_distMatrix, numPoints, d_neighborCounts);
    hipDeviceSynchronize();
    
    // Copy neighbor counts back to host for cluster expansion
    int* h_neighborCounts = (int*)malloc(numPoints * sizeof(int));
    hipMemcpy(h_neighborCounts, d_neighborCounts, numPoints * sizeof(int), hipMemcpyDeviceToHost);
    
    // Main DBSCAN algorithm
    int currentCluster = 0;
    
    for (int i = 0; i < numPoints; i++) {
        if (h_labels[i] != UNCLASSIFIED) {
            continue;  // Skip already processed points
        }
        
        if (h_neighborCounts[i] < minPts) {
            h_labels[i] = NOISE;  // Mark as noise
            continue;
        }
        
        // New cluster found, expand it
        currentCluster++;
        h_labels[i] = currentCluster;
        
        // Use vectors to track border points that need processing
        std::vector<int> borderPoints;
        borderPoints.push_back(i);
        
        int borderIndex = 0;
        
        // Expand the cluster using both host and device
        while (borderIndex < borderPoints.size()) {
            int currentBorderSize = borderPoints.size() - borderIndex;
            int* d_borderPoints;
            bool* d_changed;
            bool h_changed = false;
            
            hipMalloc((void**)&d_borderPoints, currentBorderSize * sizeof(int));
            hipMalloc((void**)&d_changed, sizeof(bool));
            
            hipMemcpy(d_borderPoints, &borderPoints[borderIndex], 
                    currentBorderSize * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice);
            hipMemcpy(d_labels, h_labels, numPoints * sizeof(int), hipMemcpyHostToDevice);
            
            // Process current set of border points
            threadsPerBlock = 256;
            blocksPerGrid = (currentBorderSize + threadsPerBlock - 1) / threadsPerBlock;
            
            expandCluster<<<blocksPerGrid, threadsPerBlock>>>(
                d_distMatrix, numPoints, d_labels, d_neighborCounts, 
                minPts, currentCluster, d_changed, d_borderPoints, currentBorderSize);
            hipDeviceSynchronize();
            
            // Get updated labels and changed flag
            hipMemcpy(h_labels, d_labels, numPoints * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
            
            // Add new border points
            for (int j = 0; j < numPoints; j++) {
                if (h_labels[j] == currentCluster && 
                    std::find(borderPoints.begin(), borderPoints.end(), j) == borderPoints.end()) {
                    if (h_neighborCounts[j] >= minPts) {
                        borderPoints.push_back(j);
                    }
                }
            }
            
            borderIndex += currentBorderSize;
            
            hipFree(d_borderPoints);
            hipFree(d_changed);
            
            if (!h_changed) {
                break;  // No more expansion possible
            }
        }
    }
    
    // Clean up
    hipFree(d_points);
    hipFree(d_distMatrix);
    hipFree(d_labels);
    hipFree(d_neighborCounts);
    free(h_neighborCounts);
}

// Main function to run the DBSCAN clustering
int main(int argc, char** argv) {
    // Parse command line arguments (if any)
    float eps = 0.5f;  // Default epsilon value
    int minPts = 5;    // Default minimum points
    int numPoints = 10000; // Default number of points
    
    if (argc > 1) numPoints = atoi(argv[1]);
    if (argc > 2) eps = atof(argv[2]);
    if (argc > 3) minPts = atoi(argv[3]);
    
    printf("Running DBSCAN with params: numPoints=%d, eps=%.2f, minPts=%d\n", 
           numPoints, eps, minPts);
    
    // Allocate host memory for points and labels
    Point3D* h_points = (Point3D*)malloc(numPoints * sizeof(Point3D));
    int* h_labels = (int*)malloc(numPoints * sizeof(int));
    
    // Generate random points for testing
    srand(42);  // Fixed seed for reproducibility
    for (int i = 0; i < numPoints; i++) {
        h_points[i].x = ((float)rand() / RAND_MAX) * 100.0f;
        h_points[i].y = ((float)rand() / RAND_MAX) * 100.0f;
        h_points[i].z = ((float)rand() / RAND_MAX) * 100.0f;
    }
    
    // Record start time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Run DBSCAN
    dbscan(h_points, numPoints, eps, minPts, h_labels);
    
    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Count clusters
    int maxCluster = 0;
    int noiseCount = 0;
    
    for (int i = 0; i < numPoints; i++) {
        if (h_labels[i] > maxCluster) {
            maxCluster = h_labels[i];
        }
        if (h_labels[i] == NOISE) {
            noiseCount++;
        }
    }
    
    printf("DBSCAN completed in %.2f ms\n", milliseconds);
    printf("Found %d clusters and %d noise points\n", maxCluster, noiseCount);
    
    // Clean up
    free(h_points);
    free(h_labels);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}